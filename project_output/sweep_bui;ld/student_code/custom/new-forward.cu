#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH 32

#define BLOCK_SIZE 32


// Optimization 1
__global__ void conv_forward_kernel_shared_mem(float *y, const float *x, const float *k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    #define y4d(i3, i2, i1, i0) y[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
    #define x4d(i3, i2, i1, i0) x[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
    #define k4d(i3, i2, i1, i0) k[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    const int H_out = H - K + 1;
    const int W_out = W - K + 1;

    const int W_grid = ceil(1.0 * W_out / TILE_WIDTH);
    const int H_grid = ceil(1.0 * H_out / TILE_WIDTH);

    extern __shared__ float sArray[];
    float * sharedX = &sArray[0];
    int twk = TILE_WIDTH + K - 1;
    float * sharedKernel = &sArray[twk * twk];

    int n = blockIdx.x;
    int m = blockIdx.y;
    int h = (blockIdx.z / W_grid) * TILE_WIDTH + threadIdx.y;
    int w = (blockIdx.z % W_grid) * TILE_WIDTH + threadIdx.x;

    int base_h = (blockIdx.z / W_grid) * TILE_WIDTH;
    int base_w = (blockIdx.z % W_grid) * TILE_WIDTH;
    
    float acc = 0;

    for (int c = 0; c < C; c++) {

        if (threadIdx.x < K && threadIdx.y < K ) {
            sharedKernel[threadIdx.y*K + threadIdx.x] = k4d(m,c,threadIdx.y,threadIdx.x);
        }

        __syncthreads();

        for (int i = h; i < base_h + twk; i+=TILE_WIDTH) {
            for (int j = w; j < base_w + twk; j+=TILE_WIDTH) {
                if (i < H && j < W) {
                    sharedX[(i - base_h) * twk + (j - base_w)] = x4d(n, c, i,j);
                } else {
                    sharedX[(i - base_h) * twk + (j - base_w)] = 0;
                }

            }

        }

        __syncthreads();

        for (int p = 0; p < K; p++) {
            for (int q = 0; q < K; q++) {
                if (threadIdx.y + p < twk && threadIdx.x + q < twk) {
                    acc += sharedX[(threadIdx.y + p) * twk + (threadIdx.x + q)] * sharedKernel[p * K + q]; 
                }    
            }
        }

        __syncthreads();

    }
    
    if (n < B && m < M && h < H_out && w < W_out) {

        y4d(n,m,h,w) = acc;
    }

    #undef y4d
    #undef x4d
    #undef k4d
    
}

// Optimization 2

__global__ void unroll_kernel(const float *x, float *unroll_x, int b, const int B, const int C, const int H, const int W, const int K, int H_out, int W_out, int unroll_H, int unroll_W) {
    
    #define x4d(i3, i2, i1, i0) x[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
      
    int idx1 = blockIdx.x * blockDim.x + threadIdx.x;
    int idx2 = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (idx1 < C * unroll_W && idx2 < (B - b)) {

        int c = idx1 / unroll_W;
        int h_out = (idx1 % unroll_W) / W_out;
        int w_out = (idx1 % unroll_W) % W_out;
        
        for (int p = 0; p < K; p++) {
            for (int q = 0; q < K; q++) {
                int h_un = c * K * K + p * K + q;
                int w_un = h_out * W_out + w_out;
                // X_unroll(idx2, h_un, w_un) = x4d(idx2 + b, c, h_out + p, w_out + q);
                unroll_x[(idx2) * (unroll_H * unroll_W) + (h_un) * (H_out * W_out) + w_un] = x4d(idx2 + b, c, h_out + p, w_out + q);
            }
        }
    }
    #undef x4d
}

__global__ void matrixMultiplyShared(float *unrolled_x, float *y, const float *k, int b, const int B, const int M, const int C, const int H, const int W, const int K, int H_out, int W_out, int unroll_H, int unroll_W) 
{

__shared__ float subTileA[TILE_WIDTH][TILE_WIDTH];
__shared__ float subTileB[TILE_WIDTH][TILE_WIDTH];

int bx = blockIdx.x;
int by = blockIdx.y;
int bz = blockIdx.z;
int tx = threadIdx.x;
int ty = threadIdx.y;
int tz = threadIdx.z;

int row = by * TILE_WIDTH + ty;
int col = bx * TILE_WIDTH + tx;
int depth = bz * blockDim.z + tz;

int h_bound = unroll_H;

float Cvalue = 0;

if (depth < B) {

    for (int q = 0; q < ceil(1.0 * (h_bound) /  TILE_WIDTH); q++) {

        if (q * TILE_WIDTH + tx < unroll_H && row < M ) {
            subTileA[ty][tx] = k[row * unroll_H + q * TILE_WIDTH + tx];
        } else {
            subTileA[ty][tx] = 0.0;
        }

        if ((q * TILE_WIDTH + ty) < unroll_H && col < unroll_W ) {
            subTileB[ty][tx] = unrolled_x[(depth * unroll_W * C * K * K) + (q * TILE_WIDTH + ty) * unroll_W + col];
        } else {
            subTileB[ty][tx] = 0.0;
        }

        __syncthreads();

        for (int j = 0; j < TILE_WIDTH; j++) {
            Cvalue += subTileA[ty][j] * subTileB[j][tx];
        }
        
        __syncthreads();

        if (row < M && col < unroll_W && depth < B - b) {
            y[ (depth + b) * M * unroll_W + row * unroll_W + col] = Cvalue;
        }
    }
}
}

// OPTIMIZATION 3 - FUSED Kernel for unrolling + matrix multiplication



// OPTIMIZATION 4 - Sweeping parameters
__global__ void sweep_kernel(float *y, const float *x, const float *k, const int B, const int M, const int C, const int H, const int W, const int K)
{

    const int H_out = H - K + 1;
    const int W_out = W - K + 1;

    const int W_grid = ceil(1.0 * W_out / TILE_WIDTH);
    const int H_grid = ceil(1.0 * H_out / TILE_WIDTH);

#define y4d(i3, i2, i1, i0) y[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
#define x4d(i3, i2, i1, i0) x[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
#define k4d(i3, i2, i1, i0) k[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]


    int n = blockIdx.x;
    int m = blockIdx.y;
    int h = (blockIdx.z / W_grid) * TILE_WIDTH + threadIdx.y;
    int w = (blockIdx.z % W_grid) * TILE_WIDTH + threadIdx.x;
    
    float acc = 0;

    if (h < H_out && w < W_out) {
        for (int c = 0; c < C; c++) {
            for (int p = 0; p < K; p++) {
                for (int q = 0; q < K; q++) {
                        acc += x4d(n, c, h+p, w+q) * k4d(m,c,p,q);
                }
            }
        }
        y4d(n,m,h,w) = acc;
    }

#undef y4d
#undef x4d
#undef k4d
}


// BASELINE
__global__ void conv_forward_kernel(float *y, const float *x, const float *k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    y - output
    x - input
    k - kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    */

    const int H_out = H - K + 1;
    const int W_out = W - K + 1;

    const int W_grid = ceil(1.0 * W_out / TILE_WIDTH);
    const int H_grid = ceil(1.0 * H_out / TILE_WIDTH);

    // (void)H_out; // silence declared but never referenced warning. remove this line when you start working
    // (void)W_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = y4d(0,0,0,0)
    // y4d(0,0,0,0) = a

#define y4d(i3, i2, i1, i0) y[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
#define x4d(i3, i2, i1, i0) x[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
#define k4d(i3, i2, i1, i0) k[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here
    // int n,m,h,w,c,p,q;

    int n = blockIdx.x;
    int m = blockIdx.y;
    int h = (blockIdx.z / W_grid) * TILE_WIDTH + threadIdx.x;
    int w = (blockIdx.z % W_grid) * TILE_WIDTH + threadIdx.y;
    
    float acc = 0;

    if (h < H_out && w < W_out) {
        for (int c = 0; c < C; c++) {
            for (int p = 0; p < K; p++) {
                for (int q = 0; q < K; q++) {
                    // if (h + p >= 0 && h + p < H && w + q >= 0 && w + q < W) {
                        acc += x4d(n, c, h+p, w+q) * k4d(m,c,p,q);
                    // }
                }
            }
        }
        y4d(n,m,h,w) = acc;
    }

#undef y4d
#undef x4d
#undef k4d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_y, const float *host_x, const float *host_k, float **device_y_ptr, float **device_x_ptr, float **device_k_ptr, const int B, const int M, const int C, const int H, const int W, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    const int H_out = H - K + 1;
    const int W_out = W - K + 1;
    
    // int Z = H_out * W_out;
    
    int sizeX = B * C * H * W * sizeof(float);
    int sizeY = B * M * H_out * W_out * sizeof(float);
    int sizeK = M * C * K * K * sizeof(float);

    hipMalloc((void **) device_y_ptr, sizeY);
    hipMalloc((void **) device_x_ptr, sizeX);
    hipMalloc((void **) device_k_ptr, sizeK); 

    // int unroll_H = C * K * K;
    // int unroll_W = H_out * W_out;

    // hipMalloc((void**) &unrolled_x, unroll_H * unroll_W * sizeof(float));

    hipMemcpy(*device_y_ptr, host_y, sizeY, hipMemcpyHostToDevice);
    hipMemcpy(*device_x_ptr, host_x, sizeX, hipMemcpyHostToDevice);
    hipMemcpy(*device_k_ptr, host_k, sizeK, hipMemcpyHostToDevice);

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }

}


__host__ void GPUInterface::conv_forward_gpu(float *device_y, const float *device_x, const float *device_k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    // Set the kernel dimensions and call the kernel
    const int H_out = H - K + 1;
    const int W_out = W - K + 1;

    const int W_grid = ceil(1.0 * W_out / TILE_WIDTH);
    const int H_grid = ceil(1.0 * H_out / TILE_WIDTH);

    int Z = H_grid * W_grid;

    dim3 blockDim(TILE_WIDTH,TILE_WIDTH,1);
    // int X = M * W_out;
    // int Y = B * H_out;
    // dim3 blockDim(X,Y,1);
    dim3 gridDim(B,M,Z);

    // BASELINE
    // conv_forward_kernel<<<gridDim, blockDim>>>(device_y,device_x,device_k,B,M,C,H,W,K);

    // OPTIMIZATION 1
    // size_t shared_size = sizeof(float) * ((TILE_WIDTH + K - 1) * (TILE_WIDTH + K - 1) + K * K);
    // conv_forward_kernel_shared_mem<<<gridDim, blockDim, shared_size>>>(device_y,device_x,device_k,B,M,C,H,W,K);

    // OPTIMIZATION 2 
    // float *unrolled_x;
    // int unroll_H = C * K * K;
    // int unroll_W = H_out * W_out;

    // int b_12 = B / 2;

    // hipMalloc((void**) &unrolled_x, b_12 * unroll_H * unroll_W * sizeof(float));

    // dim3 blockDim1(TILE_WIDTH,TILE_WIDTH,1);
    // dim3 gridDim1(ceil((float) C * H_out * W_out / TILE_WIDTH), ceil((float) b_12 / TILE_WIDTH), 1);

    // dim3 blockDim2(TILE_WIDTH,TILE_WIDTH,1);
    // dim3 gridDim2(ceil((float) unroll_W / TILE_WIDTH), ceil((float) M / TILE_WIDTH), b_12);

    // for (int b = 0; b < B; b++) {
    //     unroll_kernel<<<gridDim1, blockDim1>>>(device_x, unrolled_x,b,C,H,W,K,H_out, W_out,unroll_H, unroll_W );
    //     matrixMultiplyShared<<<gridDim2, blockDim2>>>(unrolled_x, device_y, device_k, b, M, C, H, W, K, H_out, W_out,unroll_H, unroll_W );
    // }

    // unroll_kernel<<<gridDim1, blockDim1>>>(device_x, unrolled_x,  0, b_12, C, H,  W, K,  H_out,  W_out,  unroll_H,  unroll_W);
    // matrixMultiplyShared<<<gridDim2, blockDim2>>>(unrolled_x, device_y, device_k, 0,B, M, C, H, W, K, H_out, W_out,unroll_H, unroll_W );

    // hipFree(unrolled_x);

    // hipMalloc((void**) &unrolled_x, b_12 * unroll_H * unroll_W * sizeof(float));

    // unroll_kernel<<<gridDim1, blockDim1>>>(device_x, unrolled_x,b_12,B,C,H,W,K,H_out, W_out,unroll_H, unroll_W );
    // matrixMultiplyShared<<<gridDim2, blockDim2>>>(unrolled_x, device_y, device_k, b_12,B, M, C, H, W, K, H_out, W_out,unroll_H, unroll_W );

    // hipFree(unrolled_x);

    // OPTIMIZATION 4
    sweep_kernel<<<gridDim, blockDim>>>(device_y,device_x,device_k,B,M,C,H,W,K);
    

}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_y, float *device_y, float *device_x, float *device_k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    // Copy the output back to host
    const int H_out = H - K + 1;
    const int W_out = W - K + 1;

    int sizeY = B * M * H_out * W_out * sizeof(float);

    hipMemcpy(host_y, device_y, sizeY, hipMemcpyDeviceToHost);

    hipFree(device_y);
    hipFree(device_x);
    hipFree(device_k);

    // Free device memory

}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
